#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#define MAXTHREADS 128
#define MAXBLOCKS 64

/* A bitwise function to determine the maximum exponent x that satisfies the
 * inequality 2^x < n.
 */
int floorLog2(unsigned int n) {
  int pos = 0;
  if (n >= 1<<16) { n >>= 16; pos += 16; }
  if (n >= 1<< 8) { n >>=  8; pos +=  8; }
  if (n >= 1<< 4) { n >>=  4; pos +=  4; }
  if (n >= 1<< 2) { n >>=  2; pos +=  2; }
  if (n >= 1<< 1) {           pos +=  1; }
  return ((n == 0) ? (-1) : pos);
}

/* A bitwise function to determine the minimum number n that satisfies the
 * inequality n > x, where n = 2^a for arbitrary a.
 */
unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

/* The GPU kernel that performs the power-of-two maximum value search
 * algorithm.
 */
__global__ void entrySearch_max_kernel(int *_gin, int *_gout,
  int N)
{
    // create shared memory
    extern __shared__ int _sA[];

    // load shared mem
    int tid = threadIdx.x;                              // thread index
    int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;  // A index
 
    // the larger of the two values gets written to shared memory
    if(i + blockDim.x < N) {                // if within blocked area
      if(_gin[i] > _gin[i + blockDim.x]) {
        _sA[tid] = _gin[i];
      } else {
        _sA[tid] = _gin[i + blockDim.x];
      }
    } else if (i < N) {                     // if not power of two
      _sA[tid] = _gin[i];
    } else {                                // if outside array
      _sA[tid] = 0;
    }

    __syncthreads();  // wait for all threads to catch up

    // do comparison in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
      if(tid < s) {
        if(_sA[tid] < _sA[tid + s]) {
          _sA[tid] = _sA[tid + s];
        }
      }
      __syncthreads();  // wait for all threads to catch up
    }
  
    // write result for this block to global mem
    if(tid == 0) {
      _gout[blockIdx.x] = _sA[0];
    }
}

/* The base function of the maximum search algorithm. */
int find_max(int N, int *_A)
{
  int nthreads = 0;                   // number of threads per block
  if(N < MAXTHREADS * 2) {
    nthreads = nextPow2((N+1)/2);
  } else {
    nthreads = MAXTHREADS;
  }
  int nblocks = (N + (nthreads * 2 - 1)) / (nthreads * 2);  // number of blocks

  // create temporary search array on device
  int *_tmp;
  hipMalloc((void**)&_tmp, nblocks * sizeof(int));

  dim3 dimBlocks(nthreads, 1, 1);         // the dimension of a thread block
  dim3 numBlocks(nblocks, 1, 1);          // the layout of thread blocks
  int smemSize = nthreads * sizeof(int);  // amount of shared memory

  // run kernel
  entrySearch_max_kernel<<<numBlocks, dimBlocks, smemSize>>>(_A, _tmp, N);

  hipDeviceSynchronize();  // wait until all threads catch up

  // if there was more than one block, re-run the kernel on the minimum values 
  // from each of the blocks
  while(nblocks > 1) {
    // use only the first block_number indices in min_arr
    N = nblocks;

    // recalculate kernel launch parameters
    if(N < MAXTHREADS * 2) {
      nthreads = nextPow2((N+1)/2);
    } else {
      nthreads = MAXTHREADS;
    }
    nblocks = (N + (nthreads * 2 - 1)) / (nthreads * 2);

    // run kernel
    entrySearch_max_kernel<<<numBlocks, dimBlocks, smemSize>>>(_tmp, _tmp, N);

    hipDeviceSynchronize();  // wait until all threads catch up
  }

  // copy back final answer, which resides in position zero
  int max;
  hipMemcpy(&max, _tmp, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(_tmp);
  return max;
}

/* The main test function that creates a test array of random values and calls
 * find_min(...).  It displays both the known result as maintained through the
 * CPU-generated array and the GPU test result.
 */
int main(int argc, char** argv) 
{
  int j = 0;
  double wT = 0;
  int repeat = 100;

/**** SEARCH ****/
  int dev = 0;
  int i = 0;
  int N = 0;
  int* A;
  int* _A;
  int result = 0;

  // set CUDA device
  hipSetDevice(dev);
/****************/

  // read A.dat
  printf("Reading A.dat...");
  fflush(stdout);
  FILE *ifile = fopen("A.dat", "r");
  int scan = fscanf(ifile, "%d\n", &N);
  A = (int*) malloc(N * sizeof(int));
  for(i = 0; i < N; i++) {
    scan = fscanf(ifile, "%d\n", &A[i]);
  }
  printf("done\n");
  fflush(stdout);

/**** SEARCH ****/
  // copy host array to device
  hipMalloc((void**)&_A, N * sizeof(int));
  hipMemcpy(_A, A, N * sizeof(int), hipMemcpyHostToDevice);
/****************/

  wT = omp_get_wtime();
  for(j = 0; j < repeat; j++) {
/**** SEARCH ****/
    result = find_max(N, _A);   // search
/****************/
  }
  wT = omp_get_wtime() - wT;
  printf("\nGPU:    searching A %d times...", repeat);
  fflush(stdout);
  printf("done\n        found A_max = %d in %f seconds.\n", result, wT);
  fflush(stdout);

/**** SEARCH ****/
  hipFree(_A);  // clean up device array
/****************/
  free(A);

  hipDeviceReset();
}
